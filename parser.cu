#include "hip/hip_runtime.h"

// =============================== //
//              Imports            //
// =============================== //

import cobre.system {
  void print (string);
  void quit (int);
  string readall (string);
}

import cobre.string {
  string itos (int);
}

import cobre.array (int) {
  type array as IntArr;
  IntArr new (int, int) as IntArrNew;
  int get (IntArr, int) as IntArrGet;
  void set (IntArr, int, int) as IntArrSet;
  int len (IntArr) as IntArrLen;
}

import cobre.record (string, string, IntArr) {
  type `` as Node;
  Node new (string, string, IntArr) as newNode;
  string get0(Node) as getType;
  string get1(Node) as getVal;
  IntArr get2(Node) as getChildren;
}

import cobre.array (Node) {
  type array as NodeArr;
  NodeArr new (Node, int) as NodeArrNew;
  Node get (NodeArr, int) as NodeArrGet;
  void set (NodeArr, int, Node) as NodeArrSet;
}

import cobre.record(string, string) {
  type `` as Token;
  string get0(Token) as TkType;
  string get1(Token) as TkVal;
}

import cobre.array(Token) {
  type array as TkArr;
  Token get (TkArr, int) as TkArrGet;
  int len (TkArr) as TkArrLen;
}

import cobre.record (NodeArr, int, TkArr, int) {
  type `` as Parser;
  Parser new (NodeArr, int, TkArr, int) as _ParserNew;

  NodeArr get0 (Parser) as _ndArr;
  int get1 (Parser) as _size;

  TkArr get2 (Parser) as _tkArr;
  int get3 (Parser) as _tkPos;

  void set1 (Parser, int) as _setSize;
  void set3 (Parser, int) as _setTkPos;
}

import lexer {
  TkArr tokens (string);
}

import cobre.array(string) {
  type array as StrArr;
  StrArr new (string, int) as StrArrNew;
  string get (StrArr, int) as StrArrGet;
  void set (StrArr, int, string) as StrArrSet;
}




// =============================== //
//              Methods            //
// =============================== //

Parser ParserNew (TkArr tkarr) {
  IntArr iarr = IntArrNew(0, 0);
  NodeArr ndarr = NodeArrNew(newNode("", "", iarr), 5000);
  return _ParserNew(ndarr, 0, tkarr, 0);
}

Node getNode (Parser parser, int pos) {
  NodeArr arr = _ndArr(parser);
  Node node = NodeArrGet(arr, pos);
  return node;
}

int putNode (Parser parser, Node node) {
  NodeArr arr = _ndArr(parser);
  int pos = _size(parser);
  NodeArrSet(arr, pos, node);
  _setSize(parser, pos+1);
  return pos;
}

Token peek (Parser parser) {
  int pos = _tkPos(parser);
  TkArr arr = _tkArr(parser);
  return TkArrGet(arr, pos);
}

Token next (Parser parser) {
  int pos = _tkPos(parser);
  int tk = peek(parser);
  _setTkPos(parser, pos+1);
  return tk;
}

void assert (bool arg, string msg) {
  if (arg) {} else {
    print("Asert failed: " + msg);
    quit(1);
  }
}




// =============================== //
//            Expressions          //
// =============================== //

Node parseCall (Parser parser, int base) {
  next(parser); // Skip (
  int explist = parseMaybeExprList(parser, ")");
  assert(TkType(next(parser)) == ")", "Expected ) for function call");
  IntArr children = IntArrNew(0, 2);
  IntArrSet(children, 0, base);
  IntArrSet(children, 1, explist);
  return newNode("call", "", children);
}

bool isUnop (string ty) {
  if (ty == "-") return 0<1;
  if (ty == "!") return 0<1;
  return 0<0;
}

bool isBinop (string ty) {
  if (ty == "+") return 0<1;
  if (ty == "-") return 0<1;
  if (ty == "*") return 0<1;
  if (ty == "/") return 0<1;
  if (ty == "<") return 0<1;
  if (ty == "=") return 0<1;
  if (ty == ">") return 0<1;
  if (ty == "<=") return 0<1;
  if (ty == "==") return 0<1;
  if (ty == ">=") return 0<1;
  if (ty == "!=") return 0<1;
  return 0<0;
}

int parseBaseExpr (Parser parser) {
  Token tk = next(parser);
  string ty = TkType(tk);
  IntArr iarr = IntArrNew(0, 0);
  Node node;
  if (ty == "(") {
    int nodeId = parseExpr(parser);
    assert(TkType(next(parser)) == ")", "closing ) expected for expression");
    return nodeId;
  } else if (isUnop(ty)) {
    string op = ty;
    int base = parseBaseExpr(parser);
    IntArr children = IntArrNew(base, 1);
    node = newNode("unop", op, children);
  } else if (ty == "num") {
    node = newNode("num", TkVal(tk), iarr);
  } else if (ty == "str") {
    node = newNode("str", TkVal(tk), iarr);
  } else if (ty == "true") {
    node = newNode("true", "", iarr);
  } else if (ty == "false") {
    node = newNode("true", "", iarr);
  } else if (ty == "name") {
    node = newNode("name", TkVal(tk), iarr);
    if (TkType(peek(parser)) == "(") {
      node = parseCall(parser, putNode(parser, node));
    }
  } else {
    print("Invalid expression");
    quit(1);
  }
  return putNode(parser, node);
}

int parseExpr (Parser parser) {
  int left = parseBaseExpr(parser);
  while (isBinop(TkType(peek(parser)))) {
    string op = TkType(next(parser));
    int right = parseBaseExpr(parser);
    IntArr children = IntArrNew(0, 2);
    IntArrSet(children, 0, left);
    IntArrSet(children, 1, right);
    Node node = newNode("binop", op, children);
    left = putNode(parser, node);
  }
  return left;
}

int parseExprList (Parser parser) {
  IntArr explist = IntArrNew(0, 30);
  IntArrSet(explist, 0, parseExpr(parser));
  int count = 1;

  while (TkType(peek(parser)) == ",") {
    next(parser); // Skip ,
    IntArrSet(explist, count, parseExpr(parser));
    count = count+1;
  }

  IntArr children = IntArrNew(0, count);
  int i = 0;
  while (i < count) {
    IntArrSet(children, i, IntArrGet(explist, i));
    i = i+1;
  }

  Node node = newNode("exprlist", "", children);
  return putNode(parser, node);
}

int parseMaybeExprList (Parser parser, string end) {
  if (TkType(peek(parser)) == end) {
    Node node = newNode("exprlist", "", IntArrNew(0,0));
    return putNode(parser, node);
  } else return parseExprList(parser);
}




// =============================== //
//            Statements           //
// =============================== //

int parseNameList (Parser parser, string first, string sep, string nodename, string msg) {
  StrArr names = StrArrNew("", 10);
  StrArrSet(names, 0, first);
  int namelen = 1;

  Token tk = peek(parser);
  while (TkType(tk) == sep) {
    next(parser); // Skip sep
    tk = next(parser); // Actual name token
    assert(TkType(tk) == "name", msg);
    StrArrSet(names, namelen, TkVal(tk));
    namelen = namelen+1;
  }

  IntArr nameIds = IntArrNew(0, namelen);
  int i = 0;
  while (i < namelen) {
    Node nameNode = newNode("name", StrArrGet(names, i), IntArrNew(0,0));
    int id = putNode(parser, nameNode);
    IntArrSet(nameIds, i, id);
    i = i+1;
  }

  Node namesNode = newNode(nodename, "", nameIds);
  return putNode(parser, namesNode);
}

int, string, int parseFuncSig (Parser parser, Token first, string errmsg) {
  Token tk = first;

  // ----------------- Return types
  int typesId;
  if (TkType(tk) == "void") {
    Node namesNode = newNode("typelist", "", IntArrNew(0, 0));
    typesId = putNode(parser, namesNode);
  } else if (TkType(tk) == "name") {
    string msg = "Expected function return type";
    typesId = parseNameList(parser, TkVal(tk), ",", "typelist", msg);
  } else {
    print(errmsg);
    quit(1);
  }

  // ----------------- Function Name
  tk = next(parser);
  assert(TkType(tk) == "name", "Expected function name");
  string name = TkVal(tk);

  // ----------------- Arguments
  assert(TkType(next(parser)) == "(", "( expected in function args");

  tk = peek(parser);

  int argsId;
  if (TkType(tk) == ")") {
    next(parser); // Skip )
    Node namesNode = newNode("arglist", "", IntArrNew(0, 0));
    argsId = putNode(parser, namesNode);
  } else if (TkType(tk) == "name") {

    IntArr buf = IntArrNew(0, 30);
    int count = 0;

    repeat:
      tk = next(parser);
      assert(TkType(tk) == "name", "Expected function argument type");
      string ty = TkVal(tk);
      tk = next(parser);
      string name = "";
      if (TkType(tk) == "name") {
        name = TkVal(tk);
        tk = next(parser);
      }

      Node tynode = newNode("type", ty, IntArrNew(0,0));
      Node namenode = newNode("name", name, IntArrNew(0,0));

      IntArr children = IntArrNew(0, 2);
      IntArrSet(children, 0, putNode(parser, tynode));
      IntArrSet(children, 1, putNode(parser, namenode));

      Node node = newNode("argpart", "", children);
      IntArrSet(buf, count, putNode(parser, node));
      count = count+1;

      if (TkType(tk) == ",") goto repeat;
    end:
    assert(TkType(tk) == ")", ") expected in function args");

    IntArr children = IntArrNew(0, count);
    int i = 0;
    while (i < count) {
      IntArrSet(children, i, IntArrGet(buf, i));
      i = i+1;
    }

    Node node = newNode("arglist", "", children);
    argsId = putNode(parser, node);
  }

  return typesId, name, argsId;
}

int parseImportItems (Parser parser) {
  assert(TkType(next(parser)) == "{", "Expected opening bracket for import items");
  IntArr items = IntArrNew(0, 50);
  int itemlen = 0;

  Node node;
  Token tk = next(parser);

  repeat:
    if (TkType(tk) == "}") {
      goto end;
    }
    // ------------ Type Item
    else if (TkType(tk) == "type") {
      tk = next(parser);
      assert(TkType(tk) == "name", "Expected type name");
      string name = TkVal(tk);
      string alias = name;
      tk = next(parser);
      if (TkType(tk) == "as") {
        tk = next(parser);
        assert(TkType(tk) == "name", "Expected type alias");
        alias = TkVal(tk);
        tk = next(parser);
      }
      assert(TkType(tk) == ";", "Expected ; after imported type");

      Node aliasNode = newNode("alias", alias, IntArrNew(0,0));
      int aliasId = putNode(parser, aliasNode);

      IntArr children = IntArrNew(0, 1);
      IntArrSet(children, 0, aliasId);

      node = newNode("type", name, children);

    // --------------- Function Item
    } else {
      int typesId, argsId;
      string name, alias;

      typesId, name, argsId = parseFuncSig(parser, tk, "Invalid import item");

      // ----------------- Alias
      tk = next(parser);
      if (TkType(tk) == "as") {
        tk = next(parser);
        assert(TkType(tk) == "name", "Expected type alias");
        alias = TkVal(tk);
        tk = next(parser);
      } else {
        alias = name;
      }
      assert(TkType(tk) == ";", "Expected ; after imported function");

      Node aliasNode = newNode("alias", alias, IntArrNew(0,0));
      int aliasId = putNode(parser, aliasNode);

      IntArr children = IntArrNew(0, 3);
      IntArrSet(children, 0, typesId);
      IntArrSet(children, 1, argsId);
      IntArrSet(children, 2, aliasId);

      node = newNode("function", name, children);
    }

    int nodeId = putNode(parser, node);
    IntArrSet(items, itemlen, nodeId);
    itemlen = itemlen + 1;
    tk = next(parser);
    goto repeat;
  end:

  IntArr children = IntArrNew(0, itemlen);
  int i = 0;
  while (i < itemlen) {
    IntArrSet(children, i, IntArrGet(items, i));
    i = i+1;
  }

  node = newNode("items", "", children);
  return putNode(parser, node);
}

int parseImport (Parser parser) {
  // import keyword already consumed
  Token tk;

  // ----------- Names
  int namesNodeId;
  {
    StrArr names = StrArrNew("", 10);
    tk = next(parser);
    assert(TkType(tk) == "name", "Expected first import name");
    StrArrSet(names, 0, TkVal(tk));
    int namelen = 1;

    tk = peek(parser);
    while (TkType(tk) == ".") {
      next(parser); // Skip point
      tk = next(parser); // Actual name
      assert(TkType(tk) == "name", "Expected an import name part");
      StrArrSet(names, namelen, TkVal(tk));
      namelen = namelen+1;
    }

    IntArr nameIds = IntArrNew(0, namelen);
    int i = 0;
    while (i < namelen) {
      Node nameNode = newNode("name", StrArrGet(names, i), IntArrNew(0,0));
      int id = putNode(parser, nameNode);
      IntArrSet(nameIds, i, id);
      i = i+1;
    }

    Node namesNode = newNode("namelist", "", nameIds);
    namesNodeId = putNode(parser, namesNode);
  }

  // ------------ Arguments

  int argsNodeId;
  tk = peek(parser);
  if (TkType(tk) == "(") {
    next(parser); // Skip paren
    StrArr names = StrArrNew("", 10);
    tk = next(parser);
    assert(TkType(tk) == "name", "At least one module argument required inside parens");
    StrArrSet(names, 0, TkVal(tk));
    int namelen = 1;

    tk = next(parser);
    while (TkType(tk) == ",") {
      tk = next(parser);
      assert(TkType(tk) == "name", "Not a valid import argument");
      StrArrSet(names, namelen, TkVal(tk));
      namelen = namelen+1;
      tk = next(parser);
    }

    assert(TkType(tk) == ")", "Expected closing paren after import args");

    IntArr nameIds = IntArrNew(0, namelen);
    int i = 0;
    while (i < namelen) {
      Node nameNode = newNode("name", StrArrGet(names, i), IntArrNew(0,0));
      int id = putNode(parser, nameNode);
      IntArrSet(nameIds, i, id);
      i = i+1;
    }

    Node argsNode = newNode("arglist", "", nameIds);
    argsNodeId = putNode(parser, argsNode);
  } else {
    Node argsNode = newNode("arglist", "", IntArrNew(0,0));
    argsNodeId = putNode(parser, argsNode);
  }

  int items = parseImportItems(parser);

  IntArr children = IntArrNew(0, 3);
  IntArrSet(children, 0, namesNodeId);
  IntArrSet(children, 1, argsNodeId);
  IntArrSet(children, 2, items);

  Node node = newNode("import", "", children);
  int id = putNode(parser, node);
  return id;
}

Node parseAssignment (Parser parser, string first) {
  int nameList = parseNameList(parser, first, ",", "namelist", "Expected variable name");
  assert(TkType(next(parser)) == "=", "= expected");
  int expr = parseExpr(parser);

  IntArr children = IntArrNew(0, 2);
  IntArrSet(children, 0, nameList);
  IntArrSet(children, 1, expr);

  return newNode("assignment", "", children);
}

int parseDecl (Parser parser, string typename) {
  IntArr buf = IntArrNew(0, 30);
  int count = 0;

  repeat:
    Token tk = next(parser);
    assert(TkType(tk) == "name", "variable name expected");
    string name = TkVal(tk);
    tk = next(parser);
    int exp;
    if (TkType(tk) == "=") {
      exp = parseExpr(parser);
      tk = next(parser);
    } else {
      exp = putNode(parser, newNode("none", "", IntArrNew(0,0)));
    }
    IntArr children = IntArrNew(exp, 1);
    Node node = newNode("declpart", name, children);

    IntArrSet(buf, count, putNode(parser, node));
    count = count+1;

    if (TkType(tk) == ",") goto repeat;
  end:
  assert(TkType(tk) == ";", "; expected");

  IntArr children = IntArrNew(0, count);
  int i = 0;
  while (i < count) {
    IntArrSet(children, i, IntArrGet(buf, i));
    i = i+1;
  }

  Node node = newNode("decl", typename, children);
  return putNode(parser, node);
}

int parseBlock (Parser parser) {
  IntArr buf = IntArrNew(0, 500);
  int count = 0;

  repeat:
    if (TkType(peek(parser)) == "}") goto end;
    int stmt = parseStmt(parser);
    IntArrSet(buf, count, stmt);
    count = count+1;
    goto repeat;
  end:
  next(parser); // Skip }

  IntArr children = IntArrNew(0, count);
  int i = 0;
  while (i < count) {
    IntArrSet(children, i, IntArrGet(buf, i));
    i = i+1;
  }

  Node node = newNode("block", "", children);
  return putNode(parser, node);
}

int parseIf (Parser parser) {
  assert(TkType(next(parser)) == "(", "( expected in if condition");
  int cond = parseExpr(parser);
  assert(TkType(next(parser)) == ")", ") expected in if condition");
  int stmt = parseStmt(parser);

  int els;
  if (TkType(peek(parser)) == "else") {
    next(parser);
    els = parseStmt(parser);
  } else {
    els = putNode(parser, newNode("block", "", IntArrNew(0,0)));
  }

  IntArr children = IntArrNew(0, 3);
  IntArrSet(children, 0, cond);
  IntArrSet(children, 1, stmt);
  IntArrSet(children, 2, els);

  Node node = newNode("if", "", children);
  return putNode(parser, node);
}

int parseWhile (Parser parser) {
  assert(TkType(next(parser)) == "(", "( expected in while condition");
  int cond = parseExpr(parser);
  assert(TkType(next(parser)) == ")", ") expected in while condition");
  int stmt = parseStmt(parser);

  IntArr children = IntArrNew(0, 2);
  IntArrSet(children, 0, cond);
  IntArrSet(children, 1, stmt);

  Node node = newNode("while", "", children);
  return putNode(parser, node);
}

int parseStmt (Parser parser) {
  Token tk = next(parser);
  string ty = TkType(tk);

  Node result;

  if (ty == "{") {
    return parseBlock(parser);
  } else if (ty == "goto") {
    tk = next(parser);
    assert(TkType(tk) == "name", "Expected label name");
    result = newNode("goto", TkVal(tk), IntArrNew(0, 0));
  } else if (ty == "return") {
    int exprlist = parseMaybeExprList(parser, ";");
    IntArr children = IntArrNew(exprlist, 1);
    result = newNode("return", "", children);
  } else if (ty == "if") {
    return parseIf(parser);
  } else if (ty == "while") {
    return parseWhile(parser);
  } else if (ty == "name") {
    string first = TkVal(tk);
    ty = TkType(peek(parser));
    if (ty == ":") {
      next(parser); // skip :
      Node node = newNode("label", first, IntArrNew(0, 0));
      return putNode(parser, node);
    }
    else if (ty == "=") result = parseAssignment(parser, first);
    else if (ty == ",") result = parseAssignment(parser, first);
    else if (ty == "name") return parseDecl(parser, first);
    else if (ty == "(") {
      result = newNode("name", first,IntArrNew(0,0));
      result = parseCall(parser, putNode(parser, result));
    }
    else {
      print("Statement Expected");
      quit(1);
    }
  } else {
    print("Statement Expected");
    quit(1);
  }

  assert(TkType(next(parser)) == ";", "; expected");
  return putNode(parser, result);
}

int parseBlock (Parser parser) {

  IntArr idList = IntArrNew(0, 500);
  int stmtCount = 0;

  repeat:
    if (TkType(peek(parser)) == "}") goto end;

    int statId = parseStmt(parser);
    IntArrSet(idList, stmtCount, statId);
    stmtCount = stmtCount+1;

    goto repeat;
  end:
  next(parser); // Skip }

  IntArr children = IntArrNew(0, stmtCount);
  int i = 0;
  while (i < stmtCount) {
    IntArrSet(children, i, IntArrGet(idList, i));
    i = i+1;
  }

  Node node = newNode("block", "", children);
  return putNode(parser, node);
}

int parseTopLevel (Parser parser) {
  Token tk = next(parser);

  if (TkType(tk) == "import") { return parseImport(parser); }
  else {
    int typesId, argsId;
    string name, alias;

    typesId, name, argsId = parseFuncSig(parser, tk, "Invalid toplevel statement");

    assert(TkType(next(parser)) == "{", "Expected { for function body");

    int bodyId = parseBlock(parser);

    IntArr children = IntArrNew(0, 3);
    IntArrSet(children, 0, typesId);
    IntArrSet(children, 1, argsId);
    IntArrSet(children, 2, bodyId);

    Node node = newNode("function", name, children);
    return putNode(parser, node);
  }
}




// =============================== //
//             Interface           //
// =============================== //

IntArr, NodeArr parse (string src) {

  Parser parser = ParserNew(tokens(src));

  IntArr buf = IntArrNew(0, 300);
  int count = 0;

  repeat:
    if (TkType(peek(parser)) == "eof") goto end;
    int nodeid = parseTopLevel(parser);
    IntArrSet(buf, count, nodeid);
    count = count+1;
    goto repeat;
  end:

  IntArr stmtIds = IntArrNew(0, count);
  int i = 0;
  while (i < count) {
    IntArrSet(stmtIds, i, IntArrGet(buf, i));
    i = i+1;
  }

  int size = _size(parser);
  NodeArr nodes = NodeArrNew(newNode("", "", IntArrNew(0,0)), size);
  int i = 0;
  while (i < size) {
    NodeArrSet(nodes, i, getNode(parser, i));
    i = i+1;
  }

  return stmtIds, nodes;
}

void printNode (NodeArr nodes, int pos, string indent) {
  Node node = NodeArrGet(nodes, pos);
  print(indent + getType(node) + " " + getVal(node));
  IntArr children = getChildren(node);
  int len = IntArrLen(children);
  int i = 0;
  while (i < len) {
    int id = IntArrGet(children, i);
    printNode(nodes, id, indent + "  ");
    i = i+1;
  }
}

void main () {
  NodeArr nodes;
  IntArr stmts;

  string src = readall("../culang/lexer.cu");
  stmts, nodes = parse(src);

  int len = IntArrLen(stmts);
  print(itos(len) + " statements");
  int i = 0;
  while (i < len) {
    int id = IntArrGet(stmts, i);
    printNode(nodes, id, "");
    print("");
    i = i+1;
  }
}